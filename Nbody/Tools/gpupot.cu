#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#ifdef CUDA_5
#  include <hip/hip_runtime_api.h>
#  define CUDA_SAFE_CALL checkCudaErrors
#else
#  include <cutil.h>
#endif
#include "cuda_pointer.h"

#define NTHREAD 64 // 64, 96, 128 or 192; should be same as the one in gpunb.gpu.cu
#define NJBLOCK 28 // 8800GTS/512 has 16

#define NXREDUCE 32 // must be >NJBLOCK
#define NYREDUCE 8

#define NAN_CHECK(val) assert((val) == (val));

#define PROFILE
#ifdef PROFILE
#include <sys/time.h>
static double get_wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.e-6 * tv.tv_usec;
}
#else
static double get_wtime(){
	return 0.0;
}
#endif

static float2 float2_split(double x){
	const int shift = 20;
	float2 ret;
	x *= (1<<shift);
	double xi = (int)x;
	double xf = x - xi;
	ret.x = xi * (1./(1<<shift));
	ret.y = xf * (1./(1<<shift));
	return ret;
}

static __device__ float2 float2_accum(float2 acc, float x){
  float tmp = acc.x + x;
  acc.y -= (tmp - acc.x) - x;
  acc.x = tmp;
  return acc;
}

static  __device__ float2 float2_regularize(float2 acc){
  float tmp = acc.x + acc.y;
  acc.y = acc.y -(tmp - acc.x);
  acc.x = tmp;
  return acc;
}

static __device__ float2 float2_add(float2 a, float2 b){
  float tmp = a.x + b.x;
  a.y -= (tmp - a.x) - b.x - b.y;
  a.x = tmp;
  // a.x = a.x + b.x;
  // a.y = a.y + b.y;
  return a;
}

//from gpunb.gpu.cu=================================//

struct Particle{
	float2 pos[3];
	float mass;
	float pad;

	Particle(double x[3], double m){
		pos[0] = float2_split(x[0]);
		pos[1] = float2_split(x[1]);
		pos[2] = float2_split(x[2]);
		mass = (float)m;

        NAN_CHECK(x[0]);
        NAN_CHECK(x[1]);
        NAN_CHECK(x[2]);
        NAN_CHECK(m);
	}

    Particle(double x1, double x2, double x3, double m){
		pos[0] = float2_split(x1);
		pos[1] = float2_split(x2);
		pos[2] = float2_split(x3);
		mass = (float)m;

        NAN_CHECK(x1);
        NAN_CHECK(x2);
        NAN_CHECK(x3);
        NAN_CHECK(m);
	}
    
	Particle(int){
		pos[0].x = pos[0].y = pos[1].x = pos[1].y = pos[2].x = pos[2].y = mass = pad = 0.f;
	}
	__device__ Particle() {}
};

__global__ void pot_reduce_kernel(
		const int ni,
		const float2 phipart[][NJBLOCK],
        float2 phi[]){
  //thread x * y + block x============================//
  //thread x for NJBLOCK==============================//
	const int xid = threadIdx.x;
	const int yid = threadIdx.y;
	const int bid = blockIdx.x;
    //thread y & block x for active particle============//
	const int iaddr = yid + blockDim.y * bid;

	__shared__ float2 phishare[NYREDUCE][NXREDUCE];

    __syncthreads();
	if(xid < NJBLOCK){
      phishare[yid][xid] = phipart[iaddr][xid];
	}else{
      phishare[yid][xid] = make_float2(0.f,0.f);
	}
    __syncthreads();
	float2 *phis = phishare[yid];
    
#if NXREDUCE==32
	if(xid < 16) phis[xid] = float2_add(phis[xid],phis[xid + 16]);
#endif
	if(xid < 8) phis[xid] = float2_add(phis[xid],phis[xid + 8]);
	if(xid < 4) phis[xid] = float2_add(phis[xid],phis[xid + 4]);
	if(xid < 2) phis[xid] = float2_add(phis[xid],phis[xid + 2]);
	if(xid < 1) phis[xid] = float2_add(phis[xid],phis[xid + 1]);
	
	if(iaddr < ni){
      phi[iaddr] = float2_regularize(phis[0]);
	}
}

__global__ void pot_kernel(
          int ni,                           
          int n,
          Particle *ipbuf,
          Particle *ptcl,
          float2 phipart[][NJBLOCK]){
	int i = NTHREAD * blockIdx.x + threadIdx.x;
    int jbid = blockIdx.y;
    int jstart = (n * (jbid  )) / NJBLOCK;
    int jend   = (n * (jbid+1)) / NJBLOCK;

    Particle ip=Particle();
    if(i<ni) ip = ipbuf[i];
	float2 phii = make_float2(0.f, 0.f);
	for(int j=jstart; j<jend; j+= NTHREAD){
      __shared__ Particle jpbuf[NTHREAD];

      __syncthreads();
      float4 *src = (float4 *)&ptcl[j];
      float4 *dst = (float4 *)jpbuf;
      dst[threadIdx.x]         = src[threadIdx.x];
      dst[NTHREAD+threadIdx.x] = src[NTHREAD+threadIdx.x];
      __syncthreads();

      if(jend-j < NTHREAD) {
#pragma unroll 4        
        for(int jj=0; jj<jend-j; jj++){
			// if(j+jj == i) continue;
			Particle &jp = jpbuf[jj];
			float dx = (jp.pos[0].x - ip.pos[0].x) + (jp.pos[0].y - ip.pos[0].y);
			float dy = (jp.pos[1].x - ip.pos[1].x) + (jp.pos[1].y - ip.pos[1].y);
			float dz = (jp.pos[2].x - ip.pos[2].x) + (jp.pos[2].y - ip.pos[2].y);
			float r2 = dx*dx + dy*dy + dz*dz;
			// if(r2==0.f) continue;
			float pij = jp.mass * rsqrtf(r2);
			// phii = float2_accum(phii, pij);
			if(r2 > 0.f) phii = float2_accum(phii, pij);
		}
      }else{
#pragma unroll 8
        for(int jj=0; jj<NTHREAD; jj++){
			// if(j+jj == i) continue;
			Particle &jp = jpbuf[jj];
			float dx = (jp.pos[0].x - ip.pos[0].x) + (jp.pos[0].y - ip.pos[0].y);
			float dy = (jp.pos[1].x - ip.pos[1].x) + (jp.pos[1].y - ip.pos[1].y);
			float dz = (jp.pos[2].x - ip.pos[2].x) + (jp.pos[2].y - ip.pos[2].y);
			float r2 = dx*dx + dy*dy + dz*dz;
			// if(r2==0.f) continue;
			float pij = jp.mass * rsqrtf(r2);
			// phii = float2_accum(phii, pij);
			if(r2 > 0.f) phii = float2_accum(phii, pij);
		}
      }
      phii = float2_regularize(phii);
	}
	phipart[i][jbid] = phii;
}

static int n;
static cudaPointer <Particle> ptcl;

extern "C" void gpupot_send(
		int nn,
		double m[],
        //		double x[][3],
        double x1[],
        double x2[],
        double x3[]){

	double t0 = get_wtime();
    n = nn;

    int numGPU=0;
    hipGetDeviceCount(&numGPU);
    assert(numGPU>0);
    hipSetDevice(0);
  
	int ng = NTHREAD * (n/NTHREAD + (n%NTHREAD ? 1 : 0));

	ptcl.allocate(ng);

	for(int i=0; i<n; i++){
      ptcl[i] = Particle(x1[i], x2[i], x3[i], m[i]);
	}
    for(int i=n; i<ng; i++){
      ptcl[i] = Particle(0);
    }
    ptcl.htod(ng);
    double t1 = get_wtime();
#ifdef PROFILE
	fprintf(stderr, "gpupot_send: %f sec\n", t1 - t0);
#endif
}

extern "C" void gpupot_send_3(
		int nn,
		double m[],
		double x[][3]){

	double t0 = get_wtime();
    n = nn;

    int numGPU=0;
    hipGetDeviceCount(&numGPU);
    assert(numGPU>0);
    hipSetDevice(0);
  
	int ng = NTHREAD * (n/NTHREAD + (n%NTHREAD ? 1 : 0));

	ptcl.allocate(ng);

	for(int i=0; i<n; i++){
      ptcl[i] = Particle(x[i], m[i]);
	}
    for(int i=n; i<ng; i++){
      ptcl[i] = Particle(0);
    }
    ptcl.htod(ng);
    double t1 = get_wtime();
#ifdef PROFILE
	fprintf(stderr, "gpupot_send: %f sec\n", t1 - t0);
#endif
}

extern "C" void gpupot_send_float(
		int nn,
		float m[],
		float x1[],
		float x2[],
		float x3[]){

	double t0 = get_wtime();
    n = nn;

    int numGPU=0;
    hipGetDeviceCount(&numGPU);
    assert(numGPU>0);
    hipSetDevice(0);
  
	int ng = NTHREAD * (n/NTHREAD + (n%NTHREAD ? 1 : 0));

	ptcl.allocate(ng);

    printf("m,x %f %f %f %f\n",m[0],x1[0],x2[0],x3[0]);

	for(int i=0; i<n; i++){
      ptcl[i] = Particle((double)x1[i], (double)x2[i], (double)x3[i], (double)m[i]);
	}
    for(int i=n; i<ng; i++){
      ptcl[i] = Particle(0);
    }
    ptcl.htod(ng);
    double t1 = get_wtime();
#ifdef PROFILE
	fprintf(stderr, "gpupot_send: %f sec\n", t1 - t0);
#endif
}

extern "C" void gpupot(
        int ni,
        int list[],
		double pot[]){

	double t0 = get_wtime();

    cudaPointer <float2[NJBLOCK]> phipart;
    cudaPointer <Particle> ibuf;
    cudaPointer <float2> phi;

	int ng = NTHREAD * (n/NTHREAD + (n%NTHREAD ? 1 : 0));
	int nig = NTHREAD * (ni/NTHREAD + (ni%NTHREAD ? 1 : 0));

	phi.allocate(nig);
    phipart.allocate(nig);
    ibuf.allocate(nig);

    for(int i=0; i<ni; i++) {
      ibuf[i] = ptcl[list[i]];
    }
    for(int i=ni; i<nig; i++) {
      ibuf[i] = Particle(0);
    }
    
    ibuf.htod(nig);

	dim3 grid(ng/NTHREAD, NJBLOCK, 1);
	dim3 threads(NTHREAD, 1, 1);

	pot_kernel <<<grid, threads>>> (ni, n, ibuf, ptcl, phipart);

    
    const int ni8 = 1 + (ni-1) / NYREDUCE;
    dim3 rgrid (ni8, 1, 1);
    dim3 rthreads(NXREDUCE, NYREDUCE, 1);
    pot_reduce_kernel <<< rgrid, rthreads >>> (ni, phipart, phi);

    phi.dtoh(ni);

    for(int i=0; i<ni; i++){
      pot[i] = (double)phi[i].x + (double)phi[i].y;
	}

    phi.free();
    phipart.free();
    ptcl.free();
    ibuf.free();

	double t1 = get_wtime();
#ifdef PROFILE
	fprintf(stderr, "gpupot: %f sec\n", t1 - t0);
#endif
}
