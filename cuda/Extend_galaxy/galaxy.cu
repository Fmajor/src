#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include "initial.h"
#include "hip/hip_runtime.h"
#include "cpu_bitmap.h"

const int xthreadsperblock=16;
const int ythreadsperblock=16;

struct element
{
  float x;
  float y;
  float m;
};

#ifdef _NOVISUAL
__global__ void initial(float *disp)
#endif
#ifdef _VISUAL
__global__ void initial(unsigned char *disp)
#endif
{
  int x=threadIdx.x+blockIdx.x*blockDim.x;
  int y=threadIdx.y+blockIdx.y*blockDim.y;
  int offset=x+y*blockDim.x*gridDim.x;
#ifdef _NOVISUAL
  disp[offset]=0;
#endif
#ifdef _VISUAL
  disp[4*offset] =0;
  disp[4*offset+1] =0;
  disp[4*offset+2] =255;
  disp[4*offset+3] =255;
#endif
}

#ifdef _NOVISUAL
__global__ void kernel(element *src, int p, float *disp)
#endif
#ifdef _VISUAL
__global__ void kernel(element *src, int p, unsigned char *disp)
#endif
{
  int x=threadIdx.x+blockIdx.x*blockDim.x;
  int y=threadIdx.y+blockIdx.y*blockDim.y;
  int offset=x+y*blockDim.x*gridDim.x;
  __shared__ float shared[xthreadsperblock][ythreadsperblock];
  shared[threadIdx.x][threadIdx.y]=src->m*pow((float)((x-src->x)*(x-src->x)+(y-src->y)*(y-src->y)+1),(float)p/2.0f);
  __syncthreads();
#ifdef _NOVISUAL
  disp[offset] +=shared[threadIdx.x][threadIdx.y];
#endif
#ifdef _VISUAL
  disp[4*offset+1] +=shared[threadIdx.x][threadIdx.y];
  disp[4*offset+2] -=shared[threadIdx.x][threadIdx.y];
#endif
  __syncthreads();
}

int main(int argc, char* argv[])
{
  pars_initial ini(".galaxy_gpu_config");
  ini.add("p","index of r",(int)-2);
  ini.add("xbin","bins for x-axis",(int)1024);
  ini.add("ybin","bins for y-axis",(int)1024);
  ini.add("Nsrc","source number",(int)70000);
  ini.add("infile","source file for plot: x y m","source.dat");
#ifdef _NOVISUAL
  ini.add("outfile","output file name","result.dat");
#endif
  ini.initial(argc,argv);

  int nsrc=ini.get<int>("Nsrc");
  int xmax=ini.get<int>("xbin");
  int ymax=ini.get<int>("ybin");
  int p=ini.get<int>("p");
  
  std::ifstream infile(ini.get<std::string>("infile").c_str());
  if (infile.is_open())
  {
    //Measure performance===============================//
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    //Begin=============================================//
    element *dev_src;
    element *source=(element*)malloc(sizeof(element)*nsrc);

#ifdef _VISUAL
    CPUBitmap map(xmax,ymax);
    unsigned char* dev_map;
#endif
#ifdef _NOVISUAL
    float *dev_map;
    float *map=(float*)malloc(sizeof(float)*xmax*ymax);
#endif

    for (int i=0;i<nsrc;i++)
    {
      infile>>source[i].x;
      infile>>source[i].y;
      infile>>source[i].m;
      if (infile.eof())
      {
        nsrc=i+1;
        std::cout<<"Warning: Input file reach end, change number of sources to "<<nsrc<<std::endl;
        break;
      }
    }
    
    //    for (int i=0;i<xmax*ymax;i++)
    //      map[i]=0;
#ifdef _VISUAL    
    hipMalloc((void**)&dev_map,map.image_size());
#endif
#ifdef _NOVISUAL
    hipMalloc((void**)&dev_map,xmax*ymax*sizeof(float));
#endif
    hipMalloc((void**)&dev_src,nsrc*sizeof(element));

    hipMemcpy(dev_src,source,nsrc*sizeof(element),hipMemcpyHostToDevice);
    free(source);

    //    hipMemcpy(dev_map,map,xmax*ymax*sizeof(float), hipMemcpyHostToDevice);
    
    dim3 grids(xmax/xthreadsperblock,ymax/ythreadsperblock);
    dim3 threads(xthreadsperblock,ythreadsperblock);

    initial<<<grids,threads>>>(dev_map);
    for (int i=0;i<nsrc;i++)
      kernel<<<grids,threads>>>(&dev_src[i],p,dev_map);

#ifdef _NOVISUAL
    hipMemcpy(map,dev_map,xmax*ymax*sizeof(float),hipMemcpyDeviceToHost);
#endif
#ifdef _VISUAL
    hipMemcpy(map.get_ptr(),dev_map,map.image_size(),hipMemcpyDeviceToHost);
#endif
    
    hipFree(dev_map);
    hipFree(dev_src);

    //cuda record time==================================//
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);
    std::cout<<"Time to generate: "<<elapsedTime<<" ms\n";
        
    //Output result=====================================//
#ifdef _NOVISUAL
    std::ofstream output(ini.get<std::string>("outfile").c_str());
    for (int j=0;j<ymax;j++)
    {
      for (int i=0;i<xmax;i++)
        output<<map[i+j*xmax]<<"\t";
      output<<std::endl;
    }
    output.close();
    free (map);
#endif
#ifdef _VISUAL
    map.display_and_exit();
#endif
    
  }
  else std::cerr<<"Error: input file unfound!\n";
  return 0;
}
