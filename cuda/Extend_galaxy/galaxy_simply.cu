#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include "initial.h"
#include "hip/hip_runtime.h"
#include "cpu_bitmap.h"

#define INF 1E-6
#define CONSTMEMMAXSIZE 65535

//#define _VISUAL

const int xthreadsperblock=32;
const int ythreadsperblock=32;

struct element
{
  float x;
  float y;
  float m;
  __device__ float hit(int tx, int ty, int p)
  {
    float dx=tx-x;
    float dy=ty-y;
    float r2=dx*dx+dy*dy;
    return m*pow(r2+1,(float)p);
  }
};

const int splitbin=CONSTMEMMAXSIZE/sizeof(element)-1;
__constant__ element s[splitbin];

#ifdef _NOVISUAL
__global__ void initial(float *disp)
#endif
#ifdef _VISUAL
__global__ void initial(unsigned char *disp)
#endif
{
  int x=threadIdx.x+blockIdx.x*blockDim.x;
  int y=threadIdx.y+blockIdx.y*blockDim.y;
  int offset=x+y*blockDim.x*gridDim.x;
#ifdef _NOVISUAL
  disp[offset]=0;
#endif
#ifdef _VISUAL
  disp[4*offset] =0;
  disp[4*offset+1] =0;
  disp[4*offset+2] =255;
  disp[4*offset+3] =255;
#endif
}

#ifdef _NOVISUAL
__global__ void kernel(int sbin, int p, float *disp)
#endif
#ifdef _VISUAL
__global__ void kernel(int sbin, int p, unsigned char *disp)
#endif
{
  int x=threadIdx.x+blockIdx.x*blockDim.x;
  int y=threadIdx.y+blockIdx.y*blockDim.y;
  int offset=x+y*blockDim.x*gridDim.x;
  for (int i=0;i<sbin;i++)
  {
    float value=s[i].hit(x,y,p);
    if (value>INF)
    {      
#ifdef _NOVISUAL
      disp[offset] +=value;
#endif
#ifdef _VISUAL
      disp[4*offset+1] +=value;
      disp[4*offset+2] -=value;
#endif
    }
  }
}

int main(int argc, char* argv[])
{
  pars_initial ini(".galaxy_gpu_config");
  ini.add("p","index of r",(float)-1.5);
  ini.add("xbin","bins for x-axis",(int)1024);
  ini.add("ybin","bins for y-axis",(int)1024);
  ini.add("Nsrc","source number",(int)70000);
  ini.add("infile","source file for plot: x y m","source.dat");
#ifdef _NOVISUAL
  ini.add("outfile","output file name","result.dat");
#endif
  ini.initial(argc,argv);

  int nsrc=ini.get<int>("Nsrc");
  int xmax=ini.get<int>("xbin");
  int ymax=ini.get<int>("ybin");
  float p=ini.get<int>("p")/2.0f;
  
  std::ifstream infile(ini.get<std::string>("infile").c_str());
  if (infile.is_open())
  {
    //Measure performance===============================//
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    //Begin=============================================//
    //allocate memory===================================//
    element *source=(element*)malloc(sizeof(element)*nsrc);
#ifdef _VISUAL
    CPUBitmap map(xmax,ymax);
    unsigned char* dev_map;
#endif
#ifdef _NOVISUAL
    float *dev_map;
    float *map=(float*)malloc(sizeof(float)*xmax*ymax);
#endif

    //input source data=================================//
    for (int i=0;i<nsrc;i++)
    {
      infile>>source[i].x;
      infile>>source[i].y;
      infile>>source[i].m;
      if (infile.eof())
      {
        nsrc=i+1;
        std::cout<<"Warning: Input file reach end, change number of sources to "<<nsrc<<std::endl;
        break;
      }
    }

    //GPU allocate memory===============================//
#ifdef _VISUAL    
    hipMalloc((void**)&dev_map,map.image_size());
#endif
#ifdef _NOVISUAL
    hipMalloc((void**)&dev_map,xmax*ymax*sizeof(float));
#endif

    //generate block and threads number=================//
    dim3 grids(xmax/xthreadsperblock,ymax/ythreadsperblock);
    dim3 threads(xthreadsperblock,ythreadsperblock);

    //initial display memory============================//
    initial<<<grids,threads>>>(dev_map);

    //constant memeory==================================//
    for (int i=0;i<nsrc/splitbin+1;i++)
    {
      int sbin=splitbin;
      if (i==nsrc/splitbin)
      {
        sbin=nsrc%splitbin;
      }
      element *ns=&source[i*splitbin];
      hipMemcpyToSymbol(HIP_SYMBOL(s),ns,sizeof(element)*sbin);

      //call kernel function==============================//
      kernel<<<grids,threads>>>(sbin,p,dev_map);
    }

    //copy cpu memory to gpu============================//
#ifdef _NOVISUAL
    hipMemcpy(map,dev_map,xmax*ymax*sizeof(float),hipMemcpyDeviceToHost);
#endif
#ifdef _VISUAL
    hipMemcpy(map.get_ptr(),dev_map,map.image_size(),hipMemcpyDeviceToHost);
#endif

    //free memory=======================================//
    hipFree(dev_map);
    free(source);
      
    //cuda record time==================================//
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);
    std::cout<<"Time to generate: "<<elapsedTime<<" ms\n";
        
    //Output result=====================================//
#ifdef _NOVISUAL
    std::ofstream output(ini.get<std::string>("outfile").c_str());
    for (int j=0;j<ymax;j++)
    {
      for (int i=0;i<xmax;i++)
        output<<map[i+j*xmax]<<"\t";
      output<<std::endl;
    }
    output.close();
    free (map);
#endif
#ifdef _VISUAL
    map.display_and_exit();
#endif
    
  }
  else std::cerr<<"Error: input file unfound!\n";
  return 0;
}
